
#include <hip/hip_runtime.h>
/*     CalculiX - A 3-dimensional finite element program                 */
/*              Copyright (C) 1998-2011 Guido Dhondt                     */
/*     This subroutine                                                   */
/*              Copyright (C) 2013-2015 Peter A. Gustafson               */
/*                                                                       */
/*     This program is free software; you can redistribute it and/or     */
/*     modify it under the terms of the GNU General Public License as    */
/*     published by the Free Software Foundation(version 2);    */
/*                                                                       */

/*     This program is distributed in the hope that it will be useful,   */
/*     but WITHOUT ANY WARRANTY; without even the implied warranty of    */ 
/*     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the      */
/*     GNU General Public License for more details.                      */

/*     You should have received a copy of the GNU General Public License */
/*     along with this program; if not, write to the Free Software       */
/*     Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.         */

#ifdef CUDACUSP

#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
// #include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>
// #include <cusp/krylov/cg_m.h>
// #include <cusp/krylov/bicg.h>
// #include <cusp/krylov/bicgstab.h>
#include <cusp/version.h>
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/multiply.h>
#include <cusp/precond/ainv.h> 
#include <iostream>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
// #include <cusp/krylov/gmres.h>
// #include <cusp/detail/format_utils.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
// #include <cusp/ell_matrix.h>
#include <time.h>

#ifdef LONGLONG
#define ITG long long
#define ITGFORMAT "lld"
#else
#define ITG int
#define ITGFORMAT "d"
#endif

// which floating point type to use
typedef ITG IndexType;
typedef double ValueType;
// typedef cusp::host_memory MemorySpace;
typedef cusp::device_memory MemorySpace;

template <typename T>
struct invsqr : public thrust::unary_function<T,T>
{
  __host__ __device__
  T operator()(const T& v) 
  {
    return T (1.0)/sqrt(v);
  }
};

template <typename T>
struct absolute : public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(T x)
  {
    return x < 0 ? -x : x;
  }
};

extern "C"
int cudacusp(double *ad, double *au, double *adb, double *aub, double *sigma, 
	     double *b, ITG *icol, ITG *irow, ITG *neq, ITG *nzs, 
	     int *symmetryflag, int *inputformat, ITG *jq, ITG *nzs3)
{
  int cuda_major =  CUDA_VERSION / 1000;
  int cuda_minor = (CUDA_VERSION % 1000) / 10;

  int thrust_major = THRUST_MAJOR_VERSION;
  int thrust_minor = THRUST_MINOR_VERSION;

  int cusp_major = CUSP_MAJOR_VERSION;
  int cusp_minor = CUSP_MINOR_VERSION;

  clock_t timeb;
  clock_t timee;

  std::cout << " Using CUDA based on CUSP CG SOLVER\n";
  std::cout << "   CUDA   v" << cuda_major   << "." << cuda_minor   << "\n";
  std::cout << "   Thrust v" << thrust_major << "." << thrust_minor << "\n";
  std::cout << "   Cusp   v" << cusp_major   << "." << cusp_minor   << "\n";


  timeb = clock();
  // Test for non zero values
  int nvals=0;
  for (int i=0; i<*neq; i++){if (ad[i]<0) nvals++;}
  if (nvals) {thrust::transform(ad, ad+*neq, ad, absolute<ValueType>());}

  /* Fill the matrix.  
     The off diagonal triangle is columnar from ccx
     irow() identifies the row within the column
     icol() identifies the number of non zeros within the column
     Move the the next column after achieving icol() within a column. */
   
  cusp::coo_matrix<int, ValueType, cusp::host_memory> A(*neq,*neq,2*(*nzs)+*neq);
  // ASSEMBLE FULL MATRIX.  No symmetric matrix defined in CUSP //
  // Scope for off-diagonal matrix assembly
  int k=*neq; 
  int l=0; 
  // This is somewhat expensive... can it be parallelized.  Attempted below.
  for (int i = 0; i < *neq; i++){
    // i acts as a column index
    A.row_indices[i] = i; 
    A.column_indices[i] = i; 
    A.values[i] = ad[i];
    for (int j = 0; j < icol[i]; j++){
      // Looping cols
      int nrow = irow[l]-1;
      A.row_indices[k] = nrow; 
      A.column_indices[k] = i; 
      A.values[k++] = au[l];
      // Symmetry
      A.row_indices[k] = i; 
      A.column_indices[k] = nrow; 
      A.values[k++] = au[l++];
    }
  }
  

// WORKING OMP BUT NOT FASTER //   // Perform a cumsum on the column index to make a conventional csr index
// WORKING OMP BUT NOT FASTER //   thrust::exclusive_scan(icol, icol+*neq+1, icol);
// WORKING OMP BUT NOT FASTER //   {// Scope
// WORKING OMP BUT NOT FASTER //     int i,j,k,nrow;
// WORKING OMP BUT NOT FASTER // #pragma omp parallel for private(i,j,k,nrow)
// WORKING OMP BUT NOT FASTER //     for (i = 0; i < *neq; i++){
// WORKING OMP BUT NOT FASTER //       // Diagonal elements
// WORKING OMP BUT NOT FASTER //       A.row_indices[i] = i; 
// WORKING OMP BUT NOT FASTER //       A.column_indices[i] = i; 
// WORKING OMP BUT NOT FASTER //       A.values[i] = ad[i];
// WORKING OMP BUT NOT FASTER //       k=*neq+icol[i]*2;
// WORKING OMP BUT NOT FASTER //       for (j = icol[i]; j < icol[i+1]; j++){
// WORKING OMP BUT NOT FASTER // 	nrow = irow[j]-1;
// WORKING OMP BUT NOT FASTER // 	A.row_indices[k] = nrow; 
// WORKING OMP BUT NOT FASTER // 	A.column_indices[k] = i; 
// WORKING OMP BUT NOT FASTER // 	A.values[k++] = au[j];
// WORKING OMP BUT NOT FASTER // 	// Symmetry
// WORKING OMP BUT NOT FASTER // 	A.row_indices[k] = i; 
// WORKING OMP BUT NOT FASTER // 	A.column_indices[k] = nrow; 
// WORKING OMP BUT NOT FASTER // 	A.values[k++] = au[j];
// WORKING OMP BUT NOT FASTER //       }
// WORKING OMP BUT NOT FASTER //     }
// WORKING OMP BUT NOT FASTER //   }

  A.sort_by_row_and_column();
  // cusp::print(A);
  cusp::hyb_matrix<int, ValueType, MemorySpace> AA;
  try {AA = A;}
  catch(std::bad_alloc &e)
    {
      std::cerr << "bad_alloc during transfer of A to GPU" << std::endl;
      exit(-1);
    }

  
  timee = clock();
  std::cout << "  Assembled stiffness matrix on CUDA device in = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";

  timeb = clock();
  // printf ("Smoothed aggregation algebraic multigrid preconditioner\n");
  // cusp::precond::aggregation::smoothed_aggregation<IndexType, ValueType, MemorySpace> MM(AA);
  printf ("Diagnonal preconditioner\n");
  cusp::precond::diagonal<ValueType, MemorySpace> MM(AA);
  // int nunsc=15;
  // printf ("Scaled bridson with %i non-zeros per row\n", nunsc);
  // cusp::precond::scaled_bridson_ainv<ValueType, MemorySpace> MM(AA, 0, nunsc);
  // printf ("Unscaled bridson with %i non-zeros per row\n", nunsc);
  // cusp::precond::bridson_ainv<ValueType, MemorySpace> MM(AA, 0, nunsc);

  timee = clock();
  std::cout << "  Preconditioning time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";
  
  // allocate storage for and copy right hand side (BB). 
  cusp::array1d<ValueType, MemorySpace> BB(*neq, 0.0);
  thrust::copy (b, b+*neq, BB.begin());

  timeb = clock();
  
  int i=50000;
  // if ((*b)<0.0){
  if (nvals){
    // Non-positive definite.  Give up quickly after spawning an answer
    // thrust::copy (ad, ad+*neq, DD.begin());
    // thrust::transform(DD.begin(), DD.end(), DD.begin(), absolute<ValueType>());
    i=0;
    printf ("There are %i negative values on the diagonal.  The attempt is abandoned.\n", nvals);
  }

  // set stopping criteria 
  // http://docs.cusp-library.googlecode.com/hg/classcusp_1_1default__monitor.html
  // ||b - A x|| <= absolute_tolerance + relative_tolerance * ||b||
  // cusp::default_monitor<ValueType> monitor(BB, i, 5e-3);
  // Abaqus uses a relative tolerance of 1e-3
  cusp::default_monitor<ValueType> monitor(BB, i, 1e-6);

  try 
    {
      // solve the linear system AA * XX = BB 
      cusp::krylov::cg(AA, BB, BB, monitor, MM); //Conjugate Gradient method
      timee = clock();
    }
  catch(std::bad_alloc &e)
    {
      std::cerr << "Couldn't solve system due to memory limits" << std::endl;
      exit(-1);
    }

  std::cout << "  CUDA iterative solver time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";

  // Copy the result to the b array
  thrust::copy (BB.begin(), BB.end(), b);

  if (monitor.converged()){
    std::cout << "Solver converged to " << monitor.relative_tolerance() << " relative tolerance";
    std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl;
  }else{
    std::cout << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
    std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl;
  }
  return 0;
}
#endif
