
#include <hip/hip_runtime.h>
/*     CalculiX - A 3-dimensional finite element program                 */
/*              Copyright (C) 1998-2015 Guido Dhondt                     */
/*     This subroutine                                                   */
/*              Copyright (C) 2013-2015 Peter A. Gustafson               */
/*                                                                       */
/*     This program is free software; you can redistribute it and/or     */
/*     modify it under the terms of the GNU General Public License as    */
/*     published by the Free Software Foundation(version 2);    */
/*                                                                       */

/*     This program is distributed in the hope that it will be useful,   */
/*     but WITHOUT ANY WARRANTY; without even the implied warranty of    */ 
/*     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the      */
/*     GNU General Public License for more details.                      */

/*     You should have received a copy of the GNU General Public License */
/*     along with this program; if not, write to the Free Software       */
/*     Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.         */

#ifdef CUDACUSP

#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
// #include <cusp/ell_matrix.h>
#include <cusp/krylov/cg.h>
// #include <cusp/krylov/cg_m.h>
// #include <cusp/krylov/bicg.h>
// #include <cusp/krylov/bicgstab.h>
// #include <cusp/krylov/gmres.h>
#include <cusp/version.h>
#include <cusp/array1d.h>
#include <cusp/precond/diagonal.h> 
// #include <cusp/precond/ainv.h> 
#include <cusp/precond/aggregation/smoothed_aggregation.h>
// #include <cusp/detail/format_utils.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
// #include <cusp/print.h>
#include <iostream>

#ifdef LONGLONG
#define ITG long long
#define ITGFORMAT "lld"
#else
#define ITG int
#define ITGFORMAT "d"
#endif

template <typename Monitor>
void report_status(Monitor& monitor)
{
  if (monitor.converged())
    {
      std::cout << "  Solver converged to " << monitor.tolerance() << " tolerance";
      std::cout << " after " << monitor.iteration_count() << " iterations";
      std::cout << " (" << monitor.residual_norm() << " final residual)" << "\n";
    }
  else
    {
      std::cout << "  Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
      std::cout << " to " << monitor.tolerance() << " tolerance ";
      std::cout << " (" << monitor.residual_norm() << " final residual)" << "\n";
    }
  std::cout <<  "\n\n";
}


// which floating point type to use
typedef double ValueType;
// typedef cusp::host_memory MemorySpace;
typedef cusp::device_memory MemorySpace;
// int global_recalc_cuda_M = 1;
// Can create pointers to precond matrices... can't transfer pointers to device and back as of 7/17/2013
// cusp::precond::bridson_ainv<ValueType, MemorySpace> *MM;
// cusp::precond::bridson_ainv<ValueType, cusp::host_memory> *M;




template <typename T>
struct invsqr : public thrust::unary_function<T,T>
{
  __host__ __device__
  T operator()(const T& v) 
  {
    return T (1.0)/sqrt(v);
  }
};

template <typename T>
struct absolute : public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(T x)
  {
    return x < 0 ? -x : x;
  }
};

extern "C"
int cudacusp_thrustassembly(double *ad, double *au, double *adb, double *aub, double *sigma, 
			    double *b, ITG *icol, ITG *irow, ITG *neq, ITG *nzs, 
			    int *symmetryflag, int *inputformat, ITG *jq, ITG *nzs3)
{
  int cuda_major =  CUDA_VERSION / 1000;
  int cuda_minor = (CUDA_VERSION % 1000) / 10;

  int thrust_major = THRUST_MAJOR_VERSION;
  int thrust_minor = THRUST_MINOR_VERSION;

  int cusp_major = CUSP_MAJOR_VERSION;
  int cusp_minor = CUSP_MINOR_VERSION;

  clock_t timeb;
  clock_t timee;

  std::cout << " Using CUDA based on CUSP CG SOLVER\n";
  std::cout << "   CUDA   v" << cuda_major   << "." << cuda_minor   << "\n";
  std::cout << "   Thrust v" << thrust_major << "." << thrust_minor << "\n";
  std::cout << "   Cusp   v" << cusp_major   << "." << cusp_minor   << "\n";


  timeb = clock();
  /* Fill the matrix.  ccx stores in modified compressesed sparse row
     format.  Instead of storing the pivot locations in icol, it
     stores the distance between pivots.  To make a conventional csr
     format, you must cumsum the icol vector. */

  ITG nvals=0;

  // Test for non zero values
  for (ITG i=0; i<*neq; i++){if (ad[i]<0) nvals++;}
  if (nvals) {thrust::transform(ad, ad+*neq, ad, absolute<ValueType>());}

  // Change to a zero based vector by subtracting 1
  thrust::transform(irow, irow+*nzs, thrust::make_constant_iterator(-1), irow, thrust::plus<ITG>());
  // Perform a cumsum on the column index to make a conventional csr index
  thrust::exclusive_scan(icol, icol+*neq+1, icol);

  // Create a set of "views" which act like pointers to existing memory.
  typedef typename cusp::array1d_view<ITG *> HostIndexArrayView;
  typedef typename cusp::array1d_view<ValueType *> HostValueArrayView;

  HostIndexArrayView row_offsets(icol, icol+*neq+1);
  HostIndexArrayView column_indices(irow, irow+*nzs);
  HostValueArrayView values(au, au+*nzs);
  // combine the three array1d_views into a csr_matrix_view
  typedef cusp::csr_matrix_view<HostIndexArrayView,HostIndexArrayView,HostValueArrayView> HostView;
  HostView A(*neq, *neq, *nzs, row_offsets, column_indices, values);
  
  // TRANSPOSE AND ADD ON HOST //
  cusp::coo_matrix<ITG, ValueType, cusp::host_memory> AT;
  {
    cusp::transpose(A,AT);
    cusp::add(A,AT,AT);
    
    // Create a diagonal matrix and add it to the A matrix
    // Store result in AT because A is just a matrix view to the original memory
    cusp::dia_matrix<ITG, ValueType, cusp::host_memory> D(*neq,*neq,*neq,1);
    D.diagonal_offsets[0]=0;
    for (ITG i=0; i<*neq; i++){D.values(i,0)=ad[i];}
    cusp::add(AT,D,AT);
    // Free DD
  }
  // Move to the device
  AT.sort_by_row_and_column();
  cusp::hyb_matrix<ITG, ValueType, MemorySpace> AA = AT;

  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // // Move to the device
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // cusp::hyb_matrix<ITG, ValueType, MemorySpace> AA = A;
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // // Bring the matrices together limiting scope as much as possible
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // {
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   cusp::hyb_matrix<ITG, ValueType, MemorySpace> AAT;
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   cusp::transpose(AA,AAT);
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   cusp::add(AA,AAT,AA);
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // } // free AAT
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // {
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   // Create a diagonal matrix and add it to the A matrix
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   cusp::dia_matrix<ITG, ValueType, MemorySpace> DD(*neq,*neq,*neq,1);
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   DD.diagonal_offsets[0]=0;
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   for (ITG i=0; i<*neq; i++){DD.values(i,0)=ad[i];}
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   cusp::add(AA,DD,AA);
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS //   // Free DD
  // TRANSPOSE AND ADD ON DEVICE: EXHAUSTS DEVICE MEMORY FOR LARGE MODELS // }

  // cusp::print(AA);

  timee = clock();
  std::cout << "  Assembled stiffness matrix on CUDA device in = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";

  timeb = clock();
  // set preconditioner
  printf ("Diagnonal preconditioner\n");
  cusp::precond::diagonal<ValueType, MemorySpace> MM(AA);
  timee = clock();
  std::cout << "  Preconditioning time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";
  
  // allocate storage for and copy right hand side (BB). 
  cusp::array1d<ValueType, MemorySpace> BB(*neq, 0);
  thrust::copy (b, b+*neq, BB.begin());
  
  // set stopping criteria 
  ITG i=50000;
  if (nvals){
    // Non-positive definite.  Give up quickly after spawning an answer
    i=0;
    printf ("There are %i negative values on the diagonal.  The attempt is abandoned.\n", nvals);
  }
  cusp::verbose_monitor<ValueType> monitor(BB, i, 1e-6);
    
  // solve the linear system AA * XX = BB 
  timeb = clock();
  cusp::krylov::cg(AA, BB, BB, monitor, MM); //Conjugate Gradient method
  timee = clock();

  std::cout << "  CUDA iterative solver time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << " seconds\n\n";

  thrust::copy (BB.begin(), BB.end(), b);

  if (!monitor.converged()){
    printf (" WARNING: Cuda Cusp did not find a solution.\n");
  }
  return 0;
}
#endif

